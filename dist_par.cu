// ----------------------------------------------------------------------------
// Distância de edição paralelo
// Para compilar: nvcc dist_par.cu -o dist_par -Wall
// Para executar: ./dist_par <nome arquivo entrada>


#include <hip/hip_runtime.h>
#include <stdio.h>

int n, m, *dist, *dev_dist, *dist_res;
char *s, *r, *dev_s, *dev_r;

void aloca()
{
  hipError_t 
    resultado_1,
    resultado_2,
    resultado_3,
    resultado_4;

  // aloca para host
  resultado_1 = hipHostMalloc((void **)&s, sizeof(char) * (n + 1), hipHostMallocDefault);
  resultado_2 = hipHostMalloc((void **)&r, sizeof(char) * (m + 1), hipHostMallocDefault);
  resultado_3 = hipHostMalloc((void **)&dist, sizeof(int) * (n + 1) * (m + 1), hipHostMallocDefault);
  resultado_4 = hipHostMalloc((void **)&dist_res, sizeof(int) * (n + 1) * (m + 1), hipHostMallocDefault);

  if (resultado_1 != hipSuccess)
  {
    printf("%s\n", hipGetErrorString(resultado_1));
    exit(1);
  } else if (resultado_2 != hipSuccess)
  {
    printf("%s\n", hipGetErrorString(resultado_2));
    exit(1);
  } else if (resultado_3 != hipSuccess)
  {
    printf("%s\n", hipGetErrorString(resultado_3));
    exit(1);
  } else if (resultado_4 != hipSuccess)
  {
    printf("%s\n", hipGetErrorString(resultado_4));
    exit(1);
  }

  // aloca para device
  resultado_1 = hipMalloc((void **)&dev_s, sizeof(char) * (n + 1));
  resultado_2 = hipMalloc((void **)&dev_r, sizeof(char) * (m + 1));
  resultado_3 = hipMalloc((void **)&dev_dist, sizeof(int) * (n + 1) * (m + 1));

  if (resultado_1 != hipSuccess)
  {
    printf("%s\n", hipGetErrorString(resultado_1));
    exit(1);
  } else if (resultado_2 != hipSuccess)
  {
    printf("%s\n", hipGetErrorString(resultado_2));
    exit(1);
  } else if (resultado_3 != hipSuccess)
  {
    printf("%s\n", hipGetErrorString(resultado_3));
    exit(1);
  }
}

void libera()
{
  hipFree(dev_s);
  hipFree(dev_r);
  hipFree(dev_dist);
  hipHostFree(s);
  hipHostFree(r);
  hipHostFree(dist);
  hipHostFree(dist_res);
}


void imprime_sequencias()
{
  int i;
  for(i=0; i<=n; i++)
  {
    printf("%c ", s[i]);  
  }
  printf("\n");
  for(i=0; i<=m; i++)
  {
    printf("%c ", r[i]);  
  }
  printf("\n");
}

void imprime_matriz()
{
  int i, j;
  printf("  ");
  for(i=0; i<=m; i++)
  {
    printf("%c ", r[i]);  
  }
  printf("\n");
  for(i=0; i<=n; i++)
  {
    printf("%c ", s[i]);
    for(j=0; j<=m; j++)
    {
      printf("%d ", dist[i*(m + 1)+j]);
    }
    printf("\n");
  }
}

void imprime_matriz_resposta()
{
  int i, j;
  for(i=0; i<=n; i++)
  {
    for(j=0; j<=m; j++)
    {
      printf("%d ", dist_res[i*(m + 1)+j]);
    }
    printf("\n");
  }
}


void inicializa(char* entrada)
{
  int i;
  // abre arquivo de entrada
  FILE *arq;
  arq = fopen(entrada, "rt");

  if (arq == NULL)
	{
		printf("\nArquivo texto de entrada não encontrado\n") ;
		exit(1) ;
	}

  // lê tamanho das sequências s e r
  fscanf(arq, "%d %d", &n, &m);
  //printf("entrada %d %d\n", n, m);

  // aloca sequencias e matrizes
  aloca();

  //inicializa sequencias
  s[0] = ' '; 
  r[0] = ' ';
  fscanf(arq, "%s", &(s[1]));
	fscanf(arq, "%s", &(r[1]));

  // fecha arquivo de entrada
  fclose(arq);

  // inicializa valores na matriz
  for(i=1; i<=n; i++) dist[i*(m+1)] = i;
  for(i=0; i<=m; i++) dist[i] = i;

  //printf("imprime sequencias\n");
  //imprime_sequencias();
  //printf("imprime matriz\n");
  //imprime_matriz();

  hipMemcpy(dev_s, s, sizeof(char)*(n+1), hipMemcpyHostToDevice);
  hipMemcpy(dev_r, r, sizeof(char)*(m+1), hipMemcpyHostToDevice);
  hipMemcpy(dev_dist, dist, sizeof(int)*(n+1)*(m+1), hipMemcpyHostToDevice);

}

__global__ 
void distancia_antidiagonal(int* aux, int passos, int n, int m, char* seq_s, char* seq_r) {
	int i = n - threadIdx.x - (blockIdx.x * 512);
	int j = passos - i;

	if(j>0 && j<=m && i>0)
  {
    int norte, noroeste, oeste, idx, t;

    idx = i*(m+1)+j;

    t = seq_s[i] == seq_r[j]? 0 : 1;
    norte = aux[idx-m-1] + 1;
    oeste = aux[idx-1] + 1;
    noroeste = aux[idx-m-2] + t;

    aux[idx] = min(norte, min(noroeste, oeste));
  }
}

int main(int argc, char **argv)
{

  if(argc != 2)
	{
		printf("O programa foi executado com argumentos incorretos.\n") ;
		printf("Uso: ./dist_seq <nome arquivo entrada>\n") ;
		exit(1) ;
	}

  // inicializa variáveis
  inicializa(argv[1]);

  // cria variáveis de evento
  hipEvent_t inicio, fim;

  // cria eventos
  hipEventCreate(&inicio);
  hipEventCreate(&fim);

  hipEventRecord(inicio, 0);
  
  // numero de antidiagonais
  int n_ad = n+m+1;
  // dimensao grid
  int n_blocks = ceil((double) n / 512);
  n_blocks = n_blocks == 0? 1: n_blocks;
  // numero de passos na antidiagonais
  int passos;

  for (passos=2; passos <= n_ad; passos++) {
    // calcular grid e blocos
    dim3 dimGrid(n_blocks);
    dim3 dimBlock(512);

    // calcula distancia na anti-diagonal
    distancia_antidiagonal<<<dimGrid,dimBlock>>>(dev_dist, passos, n, m, dev_s, dev_r);
  }

  // copia resposta de device para host
  hipMemcpy(dist_res, dev_dist, sizeof(int)*(n+1)*(m+1), hipMemcpyDeviceToHost);

  //imprime_matriz_resposta();
  printf("Distância=%d\n", dist_res[((n + 1) * (m + 1))-1]);

  hipEventRecord(fim, 0);
  hipEventSynchronize(fim);

  float tempo = 0;
  hipEventElapsedTime(&tempo, inicio, fim);

  // destrói eventos
  hipEventDestroy(inicio);
  hipEventDestroy(fim);

  // libera sequencias e matrizes
  libera();

  printf("Tempo GPU = %.2fms\n", tempo);

  return 0;
}
